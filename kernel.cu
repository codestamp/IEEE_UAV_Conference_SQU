// A C / C++ program for Prim's Minimum 
// Spanning Tree (MST) algorithm. The program is 
// for adjacency matrix representation of the graph 

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <limits.h> 
#include<stdbool.h> 
#include <cstdlib>
#include <ctime>
#include <algorithm>

// Number of vertices in the graph 
#define V 26 
#define K 4
// A utility function to find the vertex with 
// minimum key value, from the set of vertices 
// not yet included in MST 
int minKey(int key[], bool mstSet[])
{
	// Initialize min value 
	int min = INT_MAX, min_index=0;

	for (int v = 0; v < V; v++)
		if (mstSet[v] == false && key[v] < min)
			min = key[v], min_index = v;

	return min_index;
}

/*
void sortGraph(int graph[V][V]) 
{
	for (int xcord = 0; xcord<V; xcord++){
		for (int ycord = xcord + 1; ycord<V; ycord++){
			if (graph[xcord][0]>graph[ycord][0]){

				int temp = graph[xcord][1];
				int temp2 = graph[xcord][0];
				graph[xcord][0] = graph[ycord][0];
				graph[xcord][1] = graph[ycord][1];
				graph[ycord][0] = temp2;
				graph[ycord][1] = temp;
			}
		}
	}
}

// A utility function to print the 
// constructed MST stored in parent[] 
void printMST(int parent[], int n, int graph[V][V])
{
	//sortGraph(graph);
	printf("  Edge \t\tWeight\n");
	for (int i = 1; i < V; i++)
		printf("%2d - %2d \t%3d \n", parent[i], i, graph[i][parent[i]]);
}
*/
void kMstClusterPrint(int parent[], int n, int graph[V][V]) {
	//k number of clusters, let k=5
	int k[5] = { 0 };
	//int clusterGraph[k][V][V];

	//select random edges for removal
	srand(time(NULL));
	for (int i = 0; i < 5; i++) {
		k[i] = (int)rand() % 26;
		printf("%d\t", k[i]);
	}
	printf("\n\n");
	bool flag = false;
	for (int i = 1; i < V; i++) {
		for (int j = 0; j < 5; j++) 
		{
			if (i == k[j]) flag = true;
		}
		if (!flag) 
			printf("%2d - %2d \t%3d \n", parent[i], i, graph[i][parent[i]]);
		flag = false;
	}
}

// Function to construct and print MST for 
// a graph represented using adjacency 
// matrix representation 
void primMST(int graph[V][V])
{
	// Array to store constructed MST 
	int parent[V];
	// Key values used to pick minimum weight edge in cut 
	int key[V];
	// To represent set of vertices not yet included in MST 
	bool mstSet[V];

	// Initialize all keys as INFINITE 
	for (int i = 0; i < V; i++)
		key[i] = INT_MAX, mstSet[i] = false;

	// Always include first 1st vertex in MST. 
	// Make key 0 so that this vertex is picked as first vertex. 
	key[0] = 0;
	parent[0] = -1; // First node is always root of MST 

	// The MST will have V vertices 
	for (int count = 0; count < V - 1; count++)
	{
		// Pick the minimum key vertex from the 
		// set of vertices not yet included in MST 
		int u = minKey(key, mstSet);

		// Add the picked vertex to the MST Set 
		mstSet[u] = true;

		// Update key value and parent index of 
		// the adjacent vertices of the picked vertex. 
		// Consider only those vertices which are not 
		// yet included in MST 
		for (int v = 0; v < V; v++)

			// graph[u][v] is non zero only for adjacent vertices of m 
			// mstSet[v] is false for vertices not yet included in MST 
			// Update the key only if graph[u][v] is smaller than key[v] 
			if (graph[u][v] && mstSet[v] == false && graph[u][v] < key[v])
				parent[v] = u, key[v] = graph[u][v];
	}

	// print the constructed MST 
	kMstClusterPrint(parent, V, graph);
	//printMST(parent, V, graph);
}


// driver program to test above function 
int main()
{
	/* Let us create the following graph
	2 3
	(0)--(1)--(2)
	| / \ |
	6| 8/ \5 |7
	| /	 \ |
	(3)-------(4)
	9		 */
	int graph[V][V] = {
		{ 0, 4, 0, 0, 10, 0, 0, 0, 0, 9, 0, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//a
		{ 4, 0, 29, 0, 0, 0, 0, 0, 0, 0, 11, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//b
		{ 0, 29, 0, 8, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//c
		{ 0, 0, 8, 0, 6, 0, 0, 0, 0, 0, 17, 6, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//d
		{ 10, 0, 0, 0, 0, 9, 0, 0, 0, 0, 0, 0, 21, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//e
		{ 0, 0, 0, 0, 9, 0, 24, 0, 0, 0, 0, 0, 0, 12, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//f
		{ 0, 0, 0, 0, 0, 24, 0, 0, 0, 0, 0, 0, 0, 0, 16, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//g
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 10, 0, 12, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//h
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//i
		{ 9, 0, 0, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 0, 0, 26, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//j
		{ 0, 11, 0, 17, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 0, 0, 0 },//k
		{ 10, 0, 0, 6, 0, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0, 27, 0, 0, 0, 0, 0, 0, 0 },//l
		{ 0, 0, 0, 0, 0, 21, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0 },//m
		{ 0, 0, 0, 0, 0, 12, 0, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 16, 0, 0, 0, 0, 0, 0 },//n
		{ 0, 0, 0, 0, 0, 0, 16, 0, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 0, 0, 0, 17, 25, 0, 0 },//o
		{ 0, 0, 0, 0, 0, 0, 0, 12, 0, 0, 0, 0, 0, 0, 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//p
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 26, 0, 0, 0, 0, 0, 0, 0, 19, 0, 0, 0, 0, 0, 0, 0, 9 },//q
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 19, 0, 24, 0, 0, 0, 0, 0, 0, 16 },//r
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 27, 0, 0, 0, 0, 0, 24, 0, 0, 13, 0, 0, 0, 0, 0 },//s
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0, 16, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 13 },//t
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 8, 0, 0, 0, 0, 0, 13, 0, 0, 10, 0, 0, 0, 0 },//u
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 10, 0, 4, 0, 0, 23 },//v
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 17, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 0 },//w
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 25, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },//x
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 30 },//y
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 9, 16, 0, 13, 0, 23, 0, 0, 30, 0 }//z
	};
		
	// Print the solution 
	primMST(graph);

	return 0;
}
